#include "hip/hip_runtime.h"
#include "GPUModel.hpp"
//#include <helper_functions.h>
//#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

__global__ void speed_kernel(float* speed_gpu, float* n_running_gpu, float* density_factor_gpu, float* v_min_gpu, float* v_max_gpu, float* a_gpu, float* b_gpu, int n_road, float density_jam){
	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
	
	if(id >= n_road)	return;
	
    	speed_gpu[id] = v_min_gpu[id] + (v_max_gpu[id] - v_min_gpu[id]) * powf(1 - powf(n_running_gpu[id] * density_factor_gpu[id] / density_jam, a_gpu[id]), b_gpu[id]);

	return;
}



GPUModel::GPUModel(std::string filename, float dt, float t_end, float car_length, float car_distance,float v_max_global, float v_min_global)
:Model(filename, dt, t_end, car_length, car_distance, v_max_global, v_min_global){
	initialize_gpu_memory();
}


void GPUModel::initialize_gpu_memory(){
    hipError_t error;
    error = hipMalloc((void **)&density_gpu, n_road * sizeof(float));
    if (error != hipSuccess)
    {
        printf("hipMalloc density_gpu returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);
    }
    error = hipMalloc((void **)&speed_gpu, n_road * sizeof(float));
    if (error != hipSuccess)
    {
        printf("hipMalloc speed_gpu returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);
    }
    error = hipMalloc((void **)&n_running_gpu, n_road * sizeof(float));
    if (error != hipSuccess)
    {
        printf("hipMalloc n_running_gpu returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);
    }
    error = hipMalloc((void **)&density_factor_gpu, n_road * sizeof(float));
    if (error != hipSuccess)
    {
        printf("hipMalloc density_factor_gpu returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);
    }
    error = hipMalloc((void **)&v_min_gpu, n_road * sizeof(float));
    if (error != hipSuccess)
    {
        printf("hipMalloc v_min_gpu returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);
    }
    error = hipMalloc((void **)&v_max_gpu, n_road * sizeof(float));
    if (error != hipSuccess)
    {
        printf("hipMalloc v_max_gpu returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);
    }
    error = hipMalloc((void **)&a_gpu, n_road * sizeof(float));
    if (error != hipSuccess)
    {
        printf("hipMalloc a_gpu returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);
    }
    error = hipMalloc((void **)&b_gpu, n_road * sizeof(float));
    if (error != hipSuccess)
    {
        printf("hipMalloc b_gpu returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);
    }
    error = hipMemcpy(density_factor_gpu, density_factor, n_road * sizeof(float), hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        printf("hipMemcpy(density_factor_gpu, density_factor) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);
    }

    error = hipMemcpy(v_min_gpu, v_min, n_road * sizeof(float), hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        printf("hipMemcpy(v_min_gpu, v_min) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);
    }

    error = hipMemcpy(v_max_gpu, v_max, n_road * sizeof(float), hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        printf("hipMemcpy(v_max_gpu, v_max) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);
    }

    error = hipMemcpy(a_gpu, a, n_road * sizeof(float), hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        printf("hipMemcpy(a_gpu, a) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);
    }
    error = hipMemcpy(b_gpu, b, n_road * sizeof(float), hipMemcpyHostToDevice);
    if (error != hipSuccess)
    {
        printf("hipMemcpy(b_gpu, b) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);
    }
}

//Calculate the density for all roads
void GPUModel::get_speed()
{
    hipError_t error;
    error = hipMemcpy(n_running_gpu, n_running, n_road * sizeof(float), hipMemcpyHostToDevice);

    if (error != hipSuccess)
    {
        printf("hipMemcpy(n_running_gpu, n_running) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);
    }

    unsigned thread = 128;
    unsigned block  = ceil( 1.0f * n_road / thread );
    speed_kernel<<< block, thread >>>(speed_gpu, n_running_gpu, density_factor_gpu, v_min_gpu, v_max_gpu, a_gpu, b_gpu, n_road, density_jam);

    error = hipMemcpy(speed, speed_gpu, n_road * sizeof(float), hipMemcpyDeviceToHost);

    if (error != hipSuccess)
    {
        printf("hipMemcpy(n_running_gpu, n_running) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
        exit(EXIT_FAILURE);
    }
}
