#include "hip/hip_runtime.h"
#include "MultiGPUModel.hpp"
#include "GPUModel.hpp"
//#include <helper_functions.h>
//#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

MultiGPUModel::MultiGPUModel(std::string filename, float dt, float t_end, float car_length, float car_distance, float v_max_global, float v_min_global, std::vector<int> gpu_ids)
    : Model(filename, dt, t_end, car_length, car_distance, v_max_global, v_min_global)
{
    this->gpu_ids = gpu_ids;
    int n_road_gpu = n_road / gpu_ids.size();
    for (int i = 0; i < gpu_ids.size() - 1; i++)
    {
        n_road_per_gpu.push_back(n_road_gpu);
    }
    n_road_per_gpu.push_back(n_road - n_road_gpu * (gpu_ids.size() - 1));
    initialize_gpu_memory();
}

void MultiGPUModel::initialize_gpu_memory()
{
    int padding = 0;
    for (int i = 0; i < gpu_ids.size(); i++)
    {
        hipSetDevice(gpu_ids[i]);
        hipStream_t stream;
        hipStreamCreate(&stream);
        streams.push_back(stream);
        density_gpu.push_back(0);
        speed_gpu.push_back(0);
        n_running_gpu.push_back(0);
        density_factor_gpu.push_back(0);
        v_min_gpu.push_back(0);
        v_max_gpu.push_back(0);
        a_gpu.push_back(0);
        b_gpu.push_back(0);

        hipError_t error;
        error = hipMalloc((void **)&(density_gpu[i]), n_road_per_gpu[i] * sizeof(float));
        if (error != hipSuccess)
        {
            printf("hipMalloc density_gpu returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
            exit(EXIT_FAILURE);
        }
        error = hipMalloc((void **)&(speed_gpu[i]), n_road_per_gpu[i] * sizeof(float));
        if (error != hipSuccess)
        {
            printf("hipMalloc speed_gpu returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
            exit(EXIT_FAILURE);
        }
        error = hipMalloc((void **)&(n_running_gpu[i]), n_road_per_gpu[i] * sizeof(float));
        if (error != hipSuccess)
        {
            printf("hipMalloc n_running_gpu returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
            exit(EXIT_FAILURE);
        }
        error = hipMalloc((void **)&(density_factor_gpu[i]), n_road_per_gpu[i] * sizeof(float));
        if (error != hipSuccess)
        {
            printf("hipMalloc density_factor_gpu returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
            exit(EXIT_FAILURE);
        }
        error = hipMalloc((void **)&(v_min_gpu[i]), n_road_per_gpu[i] * sizeof(float));
        if (error != hipSuccess)
        {
            printf("hipMalloc v_min_gpu returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
            exit(EXIT_FAILURE);
        }
        error = hipMalloc((void **)&(v_max_gpu[i]), n_road_per_gpu[i] * sizeof(float));
        if (error != hipSuccess)
        {
            printf("hipMalloc v_max_gpu returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
            exit(EXIT_FAILURE);
        }
        error = hipMalloc((void **)&(a_gpu[i]), n_road_per_gpu[i] * sizeof(float));
        if (error != hipSuccess)
        {
            printf("hipMalloc a_gpu returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
            exit(EXIT_FAILURE);
        }
        error = hipMalloc((void **)&(b_gpu[i]), n_road_per_gpu[i] * sizeof(float));
        if (error != hipSuccess)
        {
            printf("hipMalloc b_gpu returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
            exit(EXIT_FAILURE);
        }
        error = hipMemcpy(density_factor_gpu[i], density_factor + padding, n_road_per_gpu[i] * sizeof(float), hipMemcpyHostToDevice);
        if (error != hipSuccess)
        {
            printf("hipMemcpy(density_factor_gpu, density_factor) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
            exit(EXIT_FAILURE);
        }

        error = hipMemcpy(v_min_gpu[i], v_min + padding, n_road_per_gpu[i] * sizeof(float), hipMemcpyHostToDevice);
        if (error != hipSuccess)
        {
            printf("hipMemcpy(v_min_gpu, v_min) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
            exit(EXIT_FAILURE);
        }

        error = hipMemcpy(v_max_gpu[i], v_max + padding, n_road_per_gpu[i] * sizeof(float), hipMemcpyHostToDevice);
        if (error != hipSuccess)
        {
            printf("hipMemcpy(v_max_gpu, v_max) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
            exit(EXIT_FAILURE);
        }

        error = hipMemcpy(a_gpu[i], a + padding, n_road_per_gpu[i] * sizeof(float), hipMemcpyHostToDevice);
        if (error != hipSuccess)
        {
            printf("hipMemcpy(a_gpu, a) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
            exit(EXIT_FAILURE);
        }
        error = hipMemcpy(b_gpu[i], b + padding, n_road_per_gpu[i] * sizeof(float), hipMemcpyHostToDevice);
        if (error != hipSuccess)
        {
            printf("hipMemcpy(b_gpu, b) returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
            exit(EXIT_FAILURE);
        }

        padding += n_road_per_gpu[i];
    }
}

//Calculate the density for all roads
void MultiGPUModel::get_speed()
{
    unsigned thread = 128;
    int padding = 0;
    for (int i = 0; i < gpu_ids.size(); i++)
    {
        unsigned block = ceil(1.0f * n_road_per_gpu[i] / thread);
        hipSetDevice(gpu_ids[i]);
        hipMemcpyAsync(n_running_gpu[i], n_running + padding, n_road_per_gpu[i] * sizeof(float), hipMemcpyHostToDevice, streams[i]);

        speed_kernel<<<block, thread, 0, streams[i]>>>(speed_gpu[i], n_running_gpu[i], density_factor_gpu[i], v_min_gpu[i], v_max_gpu[i], a_gpu[i], b_gpu[i], n_road_per_gpu[i], density_jam);
        hipMemcpyAsync(speed + padding, speed_gpu[i], n_road_per_gpu[i] * sizeof(float), hipMemcpyDeviceToHost, streams[i]);
        padding += n_road_per_gpu[i];
    }
    hipDeviceSynchronize();
}
